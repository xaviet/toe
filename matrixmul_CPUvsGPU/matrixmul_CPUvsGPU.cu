#include "hip/hip_runtime.h"
/*
CUDA initialized.
Device : " GeForce GT 1030 "

[512, 512] * [512, 512]
cpu time = 0.944000s 944
gpu time = 0.231000s 231


*/

#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include "assert.h"
#include "windows.h"
#include "time.h"
#include "hip/hip_runtime.h"
#include ""


#if __DEVICE_EMULATION__
bool InitCUDA(void) { return true; }
#else
int InitCUDA(void)
{
  int count = 0;
  int i = 0;
  hipGetDeviceCount(&count);
  if (count == 0) {
    fprintf(stderr, "There is no device.\n");
    return false;
  }
  hipDeviceProp_t prop;
  for (i = 0; i < count; i++) {
    if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
      if (prop.major >= 1) {
        break;
      }
    }
  }
  if (i == count) {
    fprintf(stderr, "There is no device supporting CUDA.\n");
    return false;
  }
  hipSetDevice(i);
  printf("CUDA initialized.\n");
  printf("Device : \" %s \" \n\n", prop.name);
  return(_ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount);
}
#endif

#define aW 1024
#define aH 1024
#define bW 1024
#define blocknum 16
#define threadnum 1024

typedef struct
{
  int width;
  int height;
  float *element;
}Matrix;
Matrix InitMatrix(int w, int h)
{
  Matrix t;
  t.element = (float *)malloc(w * h * sizeof(float));
  for (int i = 0; i < w*h; i++)
    t.element[i] = (float)(rand()) / RAND_MAX; // 0 to RAND_MAX;
  t.width = w;
  t.height = h;
  return t;
}
Matrix MM(Matrix a, Matrix b)
{
  Matrix t;
  t.element = (float *)malloc(a.height * b.width * sizeof(float));
  t.width = b.width;
  t.height = a.height;
  int x;
  int y;
  /*for (int i = 0; i < t.width * t.height; i++)
  {
    x = i / t.width * a.width;
    y = i - i / t.width * t.width;
    t.element[i] = 0;
    for (int k = 0; k < a.width; k++)
    {
      t.element[i] += a.element[x + k] * b.element[y + b.width * k];
    }
  }*/
  return t;
}
Matrix multiThreadsMM(Matrix matrixa, Matrix matrixb, DWORD dwNumberOfProcessors)
{
  Matrix t;
  t.element = (float *)malloc(matrixa.height * matrixb.width * sizeof(float));
  t.width = matrixb.width;
  t.height = matrixa.height;
  return t;
}
int printMatrix(Matrix* c, int h, int w)
{
  for (int i=0;i<c->height;i++)
  {
    if (i == h)
    {
      break;
    }
    for (int j = 0; j < c->width; j++)
    {
      if (j == w)
      {
        break;
      }
      printf("%16f, ", c->element[i*c->width + j]);
    }
    printf("\n");
  }
  return(0);
}

__global__ static void MatrixMul(float *ma, float *mb, float *mc, int *mp)
{
  int aw = mp[0];
  int bw = mp[2];
  int cw = mp[4];
  int ch = mp[5];
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  int i, x, y;

  for (i = bid * threadnum + tid; i < cw * ch; i += threadnum * blocknum)
  {
    x = i / cw * aw;
    y = i - i / cw * cw;
    mc[i] = 0;
    for (int k = 0; k < aw; k++)
    {
      mc[i] += ma[x + k] * mb[y + k * bw];
    }
  }
}


int main(int argc, char* argv[])
{
  srand(clock());
  int cudaCores = 0;
  if (!(cudaCores=InitCUDA())) {
    return 0;
  }
  //�������
  //int matrixa[N][N] , matrixb[N][N] , matrixc[N][N] , gpuresult[N][N] , matrixd[N][N] ;
  printf("[%d, %d] * [%d, %d]\n", aW, aH, bW, aW);
  Matrix matrixa = InitMatrix(aW, aH);
  printf("matrixa\n");
  printMatrix(&matrixa, 4, 4);
  Matrix matrixb = InitMatrix(bW, aW);
  printf("matrixb\n");
  printMatrix(&matrixb, 4, 4);
  Matrix matrixc;
  Matrix gpuresult = InitMatrix(bW, aH);

  int matrixprop[6];

  //ΪCPU�����ʱ

  //CPU�������
  int start = clock();
  matrixc = MM(matrixa, matrixb);
  int finish = clock();
  printf("\nmatrixc CPU\n");
  printMatrix(&matrixc, 4, 4);
  printf("cpu time(single thread)\t\t = %fs %d\n", (float)(finish - start) / CLOCKS_PER_SEC, finish - start);

  SYSTEM_INFO sysInfo;
  GetSystemInfo(&sysInfo);
  start = clock();
  Matrix matrixd;
  matrixd = multiThreadsMM(matrixa, matrixb, sysInfo.dwNumberOfProcessors);
  finish = clock();
  printf("\nmatrixd CPU\n");
  printMatrix(&matrixd, 4, 4);
  printf("cpu time(%4d threads)\t\t = %fs %d\n", sysInfo.dwNumberOfProcessors, (float)(finish - start) / CLOCKS_PER_SEC, finish - start);

  start = clock();
  matrixprop[0] = matrixa.width;
  matrixprop[1] = matrixa.height;
  matrixprop[2] = matrixb.width;
  matrixprop[3] = matrixb.height;
  matrixprop[4] = matrixc.width;
  matrixprop[5] = matrixc.height;

  //�����Դ�
  float *ma, *mb, *mc;
  int *mp;
  hipMalloc((void**)&ma, sizeof(float) * matrixa.width * matrixa.height);
  hipMalloc((void**)&mb, sizeof(float) * matrixb.width * matrixb.height);
  hipMalloc((void**)&mc, sizeof(float) * matrixc.width * matrixc.height);
  hipMalloc((void**)&mp, sizeof(int) * 6);
  //�����ݸ��Ƶ��Դ���
  hipMemcpy(ma, matrixa.element, sizeof(float) * matrixa.width * matrixa.height, hipMemcpyHostToDevice);
  hipMemcpy(mb, matrixb.element, sizeof(float) * matrixb.width * matrixb.height, hipMemcpyHostToDevice);
  hipMemcpy(mp, matrixprop, sizeof(int) * 6, hipMemcpyHostToDevice);
  //����CUDA����
  MatrixMul <<< blocknum, threadnum >>>(ma, mb, mc, mp);
  hipDeviceSynchronize();
  //cutilCheckError( cutStopTimer( timer2));
  //�����ݴ��Դ��и��Ƴ���
  hipMemcpy(gpuresult.element, mc, sizeof(float) * gpuresult.width * gpuresult.height, hipMemcpyDeviceToHost);
  finish = clock();
  printf("\ngpuresult GPU\n");
  printMatrix(&gpuresult, 4, 4);
  printf("gpu time(%4d CUDA Cores)\t = %fs %d\n", cudaCores, (float)(finish - start) / CLOCKS_PER_SEC, finish - start);
  float err = 0;
  for (int i = 0; i < gpuresult.width * gpuresult.height; i++)
  {
    //if (matrixc.element[i] != gpuresult.element[i])
    //{
      //printf("ERROR");
    //}
    err += matrixc.element[i] - gpuresult.element[i];
    //printf("%f - %f = %f \n", matrixc.element[i], gpuresult.element[i], err);
  }
  printf("\nerror: %f\n", err / (gpuresult.width * gpuresult.height));

  hipFree(ma);
  hipFree(mb);
  hipFree(mc);
  hipFree(mp);

  printf("\nPress any key to exit.\n");
  getchar();

  return 0;
}