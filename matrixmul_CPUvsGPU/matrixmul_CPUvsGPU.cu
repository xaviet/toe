#include "hip/hip_runtime.h"
/*
CUDA initialized.
Device : " GeForce GT 1030 "

[512, 512] * [512, 512]
cpu time = 0.944000s 944
gpu time = 0.231000s 231


*/

#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include "assert.h"
#include "windows.h"
#include "time.h"
#include "hip/hip_runtime.h"
#include ""


#if __DEVICE_EMULATION__
bool InitCUDA(void) { return true; }
#else
int InitCUDA(void)
{
  int count = 0;
  int i = 0;
  hipGetDeviceCount(&count);
  if (count == 0) {
    fprintf(stderr, "There is no device.\n");
    return false;
  }
  hipDeviceProp_t prop;
  for (i = 0; i < count; i++) {
    if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
      if (prop.major >= 1) {
        break;
      }
    }
  }
  if (i == count) {
    fprintf(stderr, "There is no device supporting CUDA.\n");
    return false;
  }
  hipSetDevice(i);
  printf("CUDA initialized.\n");
  printf("Device : \" %s \" \n\n", prop.name);
  return(_ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount);
}
#endif

#define aW 2
#define aH 2
#define bW 2
#define blocknum 32//32
#define threadnum 256//256

typedef struct
{
  int width;
  int height;
  float *element;
}Matrix;
Matrix InitMatrix(int w, int h)
{
  Matrix t;
  t.element = (float *)malloc(w * h * sizeof(float));
  for (int i = 0; i < w*h; i++)
    t.element[i] = (float)(rand()) / RAND_MAX; // 0 to RAND_MAX;
  t.width = w;
  t.height = h;
  return t;
}
Matrix MM(Matrix a, Matrix b)
{
  Matrix t;
  t.element = (float *)malloc(a.height * b.width * sizeof(float));
  t.width = b.width;
  t.height = a.height;
  int x;
  int y;
  for (int i = 0; i < t.width * t.height; i++)
  {
    x = i / t.width * a.width;
    y = i - i / t.width * t.width;
    t.element[i] = 0;
    for (int k = 0; k < a.width; k++)
    {
      t.element[i] += a.element[x + k] * b.element[y + b.width * k];
    }
  }
  return t;
}
Matrix multiThreadsMM(Matrix matrixa, Matrix matrixb, DWORD dwNumberOfProcessors)
{
  Matrix t;
  t.element = (float *)malloc(matrixa.height * matrixb.width * sizeof(float));
  t.width = matrixb.width;
  t.height = matrixa.height;
  return t;
}
int printMatrix(Matrix* c, int h, int w)
{
  for (int i=0;i<c->height;i++)
  {
    if (i == h)
    {
      break;
    }
    for (int j = 0; j < c->width; j++)
    {
      if (j == w)
      {
        break;
      }
      printf("%6f, ", c->element[i*c->width + j]);
    }
    printf("\n");
  }
  return(0);
}

__global__ static void MatrixMul(float *ma, float *mb, float *mc, int *mp)
{
  int aw = mp[0];
  int bw = mp[2];
  int cw = mp[4];
  int ch = mp[5];
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  int i, x, y;

  for (i = bid * threadnum + tid; i < cw * ch; i += threadnum * blocknum)
  {
    x = i / cw * aw;
    y = i - i / cw * cw;
    mc[i] = 0;
    for (int k = 0; k < aw; k++)
    {
      mc[i] += ma[x + k] * mb[y + k * bw];
    }
  }
}


int main(int argc, char* argv[])
{
  srand(clock());
  int cudaCores = 0;
  if (!(cudaCores=InitCUDA())) {
    return 0;
  }
  //�������
  //int matrixa[N][N] , matrixb[N][N] , matrixc[N][N] , gpuresult[N][N] , matrixd[N][N] ;
  printf("[%d, %d] * [%d, %d]\n", aW, aH, bW, aW);
  Matrix matrixa = InitMatrix(aW, aH);
  printMatrix(&matrixa, 2, 2);
  Matrix matrixb = InitMatrix(bW, aW);
  printMatrix(&matrixb, 2, 2);
  Matrix matrixc;
  Matrix gpuresult = InitMatrix(bW, aH);

  int matrixprop[6];

  //ΪCPU�����ʱ

  //CPU�������
  int start = clock();
  matrixc = MM(matrixa, matrixb);
  int finish = clock();
  printMatrix(&matrixc, 4, 4);
  printf("cpu time(single thread)\t\t = %fs %d\n", (float)(finish - start) / CLOCKS_PER_SEC, finish - start);

  SYSTEM_INFO sysInfo;
  GetSystemInfo(&sysInfo);
  start = clock();
  matrixc = multiThreadsMM(matrixa, matrixb, sysInfo.dwNumberOfProcessors);
  finish = clock();
  printf("cpu time(%4d threads)\t\t = %fs %d\n", sysInfo.dwNumberOfProcessors, (float)(finish - start) / CLOCKS_PER_SEC, finish - start);

  start = clock();
  matrixprop[0] = matrixa.width;
  matrixprop[1] = matrixa.height;
  matrixprop[2] = matrixb.width;
  matrixprop[3] = matrixb.height;
  matrixprop[4] = matrixc.width;
  matrixprop[5] = matrixc.height;

  //�����Դ�
  float *ma, *mb, *mc, *mp;
  hipMalloc((void**)&ma, sizeof(float) * matrixa.width * matrixa.height);
  hipMalloc((void**)&mb, sizeof(float) * matrixb.width * matrixb.height);
  hipMalloc((void**)&mc, sizeof(float) * matrixc.width * matrixc.height);
  hipMalloc((void**)&mp, sizeof(float) * 6);
  //�����ݸ��Ƶ��Դ���
  hipMemcpy(ma, matrixa.element, sizeof(float) * matrixa.width * matrixa.height, hipMemcpyHostToDevice);
  hipMemcpy(mb, matrixb.element, sizeof(float) * matrixb.width * matrixb.height, hipMemcpyHostToDevice);
  hipMemcpy(mp, matrixprop, sizeof(int) * 6, hipMemcpyHostToDevice);
  //����CUDA����
  MatrixMul <<< blocknum, threadnum, 0 >>>(ma, mb, mc, mp);
  hipDeviceSynchronize();
  //cutilCheckError( cutStopTimer( timer2));
  //�����ݴ��Դ��и��Ƴ���
  hipMemcpy(gpuresult.element, mc, sizeof(float) * gpuresult.width * gpuresult.height, hipMemcpyDeviceToHost);
  finish = clock();
  printf("gpu time(%4d CUDA Cores)\t = %fs %d\n", cudaCores, (float)(finish - start)/CLOCKS_PER_SEC, finish - start);
  printMatrix(&gpuresult, 4, 4);
  for (int i = 0; i < gpuresult.width * gpuresult.height; i++)
  {
    //printf("%d -- %d\n",matrixc.element[ i ],gpuresult.element[ i ]);
    if (matrixc.element[i] != gpuresult.element[i])
    {
      //printf("ERROR");
    }
  }

  hipFree(ma);
  hipFree(mb);
  hipFree(mc);
  hipFree(mp);

  printf("\nPress any key to exit.\n");
  getchar();

  return 0;
}