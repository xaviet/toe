#include "hip/hip_runtime.h"
/*
CUDA initialized.
Device : " GeForce GT 1030 "

[512, 512] * [512, 512]
cpu time = 0.944000s 944
gpu time = 0.231000s 231


*/

#include "stdio.h"
#include "assert.h"
#include "windows.h"
#include "time.h"
#include "hip/hip_runtime.h"
#include ""


#if __DEVICE_EMULATION__
bool InitCUDA(void) { return true; }
#else
bool InitCUDA(void)
{
  int count = 0;
  int i = 0;
  hipGetDeviceCount(&count);
  if (count == 0) {
    fprintf(stderr, "There is no device.\n");
    return false;
  }
  for (i = 0; i < count; i++) {
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
      if (prop.major >= 1) {
        break;
      }
    }
  }
  if (i == count) {
    fprintf(stderr, "There is no device supporting CUDA.\n");
    return false;
  }
  hipSetDevice(i);
  printf("CUDA initialized.\n");
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, i);
  printf("Device : \" %s \" \n\n", prop.name);
  return true;
}
#endif

#define aW 512
#define aH 512
#define bW 512
#define blocknum 32//32
#define threadnum 256//256

typedef struct
{
  int width;
  int height;
  int *element;
}Matrix;
Matrix InitMatrix(int w, int h)
{
  Matrix t;
  t.element = (int *)malloc(w * h * sizeof(int));
  for (int i = 0; i < w*h; i++)
    t.element[i] = rand() % 10;
  t.width = w;
  t.height = h;
  return t;
}
Matrix MM(Matrix a, Matrix b)
{
  Matrix t;
  t.element = (int *)malloc(a.height * b.width * sizeof(int));
  t.width = b.width;
  t.height = a.height;
  int x;
  int y;
  for (int i = 0; i < t.width * t.height; i++)
  {
    x = i / t.width * a.width;
    y = i - i / t.width * t.width;
    t.element[i] = 0;
    for (int k = 0; k < a.width; k++)
    {
      t.element[i] += a.element[x + k] * b.element[y + b.width * k];
    }
  }
  return t;
}


__global__ static void MatrixMul(int *ma, int *mb, int *mc, int *mp)
{
  int aw = mp[0];
  int bw = mp[2];
  int cw = mp[4];
  int ch = mp[5];
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  int i, x, y;

  for (i = bid * threadnum + tid; i < cw * ch; i += threadnum * blocknum)
  {
    x = i / cw * aw;
    y = i - i / cw * cw;
    mc[i] = 0;
    for (int k = 0; k < aw; k++)
    {
      mc[i] += ma[x + k] * mb[y + k * bw];
    }
  }
}


int main(int argc, char* argv[])
{
  if (!InitCUDA()) {
    return 0;
  }
  //�������
  //int matrixa[N][N] , matrixb[N][N] , matrixc[N][N] , gpuresult[N][N] , matrixd[N][N] ;
  printf("[%d, %d] * [%d, %d]\n", aW, aH, bW, aW);
  Matrix matrixa = InitMatrix(aW, aH);
  Matrix matrixb = InitMatrix(bW, aW);
  Matrix matrixc;
  Matrix gpuresult = InitMatrix(bW, aH);

  int matrixprop[6];

  //ΪCPU�����ʱ

  //CPU�������
  int start = clock();
  matrixc = MM(matrixa, matrixb);
  int finish = clock();
  printf("cpu time = %fs %d\n", (float)(finish - start) / CLOCKS_PER_SEC, finish - start);

  start = clock();
  matrixprop[0] = matrixa.width;
  matrixprop[1] = matrixa.height;
  matrixprop[2] = matrixb.width;
  matrixprop[3] = matrixb.height;
  matrixprop[4] = matrixc.width;
  matrixprop[5] = matrixc.height;

  //�����Դ�
  int *ma, *mb, *mc, *mp;
  hipMalloc((void**)&ma, sizeof(int) * matrixa.width * matrixa.height);
  hipMalloc((void**)&mb, sizeof(int) * matrixb.width * matrixb.height);
  hipMalloc((void**)&mc, sizeof(int) * matrixc.width * matrixc.height);
  hipMalloc((void**)&mp, sizeof(int) * 6);
  //�����ݸ��Ƶ��Դ���
  hipMemcpy(ma, matrixa.element, sizeof(int) * matrixa.width * matrixa.height, hipMemcpyHostToDevice);
  hipMemcpy(mb, matrixb.element, sizeof(int) * matrixb.width * matrixb.height, hipMemcpyHostToDevice);
  hipMemcpy(mp, matrixprop, sizeof(int) * 6, hipMemcpyHostToDevice);
  //����CUDA����
  MatrixMul <<< blocknum, threadnum, 0 >>>(ma, mb, mc, mp);
  hipDeviceSynchronize();
  //cutilCheckError( cutStopTimer( timer2));
  //�����ݴ��Դ��и��Ƴ���
  hipMemcpy(gpuresult.element, mc, sizeof(int) * gpuresult.width * gpuresult.height, hipMemcpyDeviceToHost);
  finish = clock();
  printf("gpu time = %fs %d\n", (float)(finish - start)/CLOCKS_PER_SEC, finish - start);

  for (int i = 0; i < gpuresult.width * gpuresult.height; i++)
  {
    //printf("%d -- %d\n",matrixc.element[ i ],gpuresult.element[ i ]);
    if (matrixc.element[i] != gpuresult.element[i])
    {
      printf("ERROR");
    }
  }

  hipFree(ma);
  hipFree(mb);
  hipFree(mc);
  hipFree(mp);

  printf("\nPress any key to exit.\n");
  getchar();

  return 0;
}